//
// Created by zqp on 19-7-20.
//
#include <mathFunction.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <iostream>

using namespace std;

bool Math::Dgemm(int m, int k, int n, double *a, double *b, double *c) {
    double *dev_a, *dev_b, *dev_c;
    hipblasStatus_t stat;
    hipError_t stat_a, stat_b, stat_c;

    stat_a = hipMalloc((void**)&dev_a, m*k*sizeof(double));
    stat_b = hipMalloc((void**)&dev_b, k*n*sizeof(double));
    stat_c = hipMalloc((void**)&dev_c, m*n*sizeof(double));
    if (stat_a != hipSuccess || stat_b!=hipSuccess || stat_c!=hipSuccess) {
        if(stat_a==hipSuccess)
            hipFree(dev_a);
        if(stat_b==hipSuccess)
            hipFree(dev_b);
        if(stat_c==hipSuccess)
            hipFree(dev_c);
        printf("device memory allocation failed\n");
        return false;
    }

    stat_a = hipMemcpy(dev_a, a, m*k*sizeof(double), hipMemcpyHostToDevice);
    stat_b = hipMemcpy(dev_b, b, k*n*sizeof(double), hipMemcpyHostToDevice);
    if (stat_a!=hipSuccess || stat_b!=hipSuccess) {
        printf("data download failed\n");
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        return false;
    }

    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        return false;
    }

    double alpha = 1.0;
    double beta = 0.0;

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dev_b, n, dev_a, k, &beta, dev_c, n);

    stat_c = hipMemcpy(c,dev_c,m*n*sizeof(double),hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipblasDestroy(handle);

    if(stat_c!=hipSuccess){
        printf("copy dev_c--->c failed\n");
        return false;
    }

    return true;
}

bool Math::Sgemm(int m, int k, int n, float *a, float *b, float *c) {
    float *dev_a, *dev_b, *dev_c;
    hipblasStatus_t stat;
    hipError_t stat_a, stat_b, stat_c;

    stat_a = hipMalloc((void**)&dev_a, m*k*sizeof(float));
    stat_b = hipMalloc((void**)&dev_b, k*n*sizeof(float));
    stat_c = hipMalloc((void**)&dev_c, m*n*sizeof(float));
    if (stat_a != hipSuccess || stat_b!=hipSuccess || stat_c!=hipSuccess) {
        if(stat_a==hipSuccess)
            hipFree(dev_a);
        if(stat_b==hipSuccess)
            hipFree(dev_b);
        if(stat_c==hipSuccess)
            hipFree(dev_c);
        printf("device memory allocation failed\n");
        return false;
    }

    stat_a = hipMemcpy(dev_a, a, m*k*sizeof(float), hipMemcpyHostToDevice);
    stat_b = hipMemcpy(dev_b, b, k*n*sizeof(float), hipMemcpyHostToDevice);
    if (stat_a!=hipSuccess || stat_b!=hipSuccess) {
        printf("data download failed\n");
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        return false;
    }

    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        return false;
    }

    float alpha = 1.0;
    float beta = 0.0;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, dev_b, n, dev_a, k, &beta, dev_c, n);

    stat_c = hipMemcpy(c,dev_c,m*n*sizeof(float),hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipblasDestroy(handle);

    if(stat_c!=hipSuccess){
        printf("copy dev_c--->c failed\n");
        return false;
    }

    return true;
}




